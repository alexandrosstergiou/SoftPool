#include "hip/hip_runtime.h"
#include <float.h>
#include <ATen/ATen.h>
#include <THC/THCAtomics.cuh>

#include "limits.cuh"

using namespace at;  // fix for pytorch<=0.4.1

#define CUDA_1D_KERNEL_LOOP(i, n)                            \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; \
       i += blockDim.x * gridDim.x)

#define THREADS_PER_BLOCK 1024

inline int GET_BLOCKS(const int N) {
  int optimal_block_num = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
  int max_block_num = 65000;
  return min(optimal_block_num, max_block_num);
}

//type-safe sign
template <typename scalar_t>
__device__ scalar_t sgn(scalar_t val) {
    return (scalar_t(0) < val) - (val < scalar_t(0));
}

// Overflow and Underflow clamp
template <typename scalar_t>
__device__  scalar_t clamp(const scalar_t n, const scalar_t lower, const scalar_t upper) {
  const scalar_t tmp = abs(n);
  const scalar_t result = max(lower, min(tmp, upper));
  return result * sgn(n);
}


template <typename scalar_t>
__global__ void SoftPool1dForward(const int nthreads,
                                  const scalar_t *bottom_input, const int batches,
                                  const int channels, const int dim,
                                  const int kernel_d, const int stride_d,
                                  scalar_t *output_data){
  int pooled_dim = dim/stride_d;
  // Run in parallel for each cell within each kernel region
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    int pd = index % pooled_dim;// index of each kernel operation in relation to the position in the input
    int c = (index / pooled_dim) % channels;
    int n = index / pooled_dim / channels;

    const int offset = (n * channels + c) * dim; // initial offset
    const scalar_t *offset_bottom_input = bottom_input + offset;

    const int base_d = pd*stride_d; // start cell index for each kernel
    if (base_d > dim - kernel_d)break; // limit iterations based on the position of the final kernel application over the input

    // --- Initialisations happen here ----
    scalar_t mask_sum_max = 0.;

    output_data[index] = 0.;
    const scalar_t upper = n_limits<scalar_t>::max();
    const scalar_t lower = n_limits<scalar_t>::min();
    const scalar_t zero = 0.;

    // Iterate over inputs cells within each kernel region in the input
    for(int id=0; id<kernel_d; id++){
      const int d_offset = base_d + id;

      if(d_offset >= dim || d_offset < 0)continue;// check if the offset index is valid (not larger than or equal to the size of the dimension) OR smaller than 0 (for fool proofing)
      const int offset = d_offset;

      // Use this for verbose when debugging
      //printf("(pd: %d), base_d: %d, id: %d, d_offset: %d \n", pd, base_d, id, d_offset);

      mask_sum_max += exp(offset_bottom_input[offset]);

    }
    // Overflow check
    mask_sum_max = clamp(mask_sum_max, lower, upper);

    for(int id=0; id<kernel_d; id++){
      const int d_offset = base_d + id;

      if(d_offset >= dim || d_offset < 0)continue;
      const int offset = d_offset;

      scalar_t mask_ = exp(offset_bottom_input[offset])/ mask_sum_max;// SoftMax

      output_data[index] += offset_bottom_input[offset] * mask_;
      output_data[index] = clamp(output_data[index], zero, upper);
    }
  }
}


template <typename scalar_t>
__global__ void SoftPool2dForward(const int nthreads,
                                  const scalar_t *bottom_input, const int batches,
                                  const int channels, const int height,
                                  const int width, const int kernel_h,
                                  const int kernel_w, const int stride_h,
                                  const int stride_w, scalar_t *output_data){
  int pooled_height = height/stride_h;
  int pooled_width = width/stride_w;
  // Run in parallel for each cell within each kernel region
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    int pw = index % pooled_width; // index over width of each kernel operation in relation to the position in the input
    int ph = (index / pooled_width) % pooled_height; // index  over height of each kernel operation in relation to the position in the input
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;

    const int offset = (n * channels + c) * height * width; // initial offset
    const scalar_t *offset_bottom_input = bottom_input + offset;

    const int base_y = ph*stride_h;// start cell index over height/y for each kernel
    if (base_y > height - kernel_h)break; // limit height/y iterations for the index of the final kernel location in the input

    const int base_x = pw*stride_w; // start cell index over width/x for each kernel
    if (base_x > width - kernel_w)break; // limit width/x iterations for the index of the final kernel location in the input

    // --- Initialisations happen here ----
    scalar_t mask_sum_max = 0.;

    output_data[index] = 0.;
    const scalar_t upper = n_limits<scalar_t>::max();
    const scalar_t lower = n_limits<scalar_t>::min();
    const scalar_t zero = 0.;

    // Iterate over inputs cells within each kernel region in the input
    for(int iy=0; iy<kernel_h; iy++){
      const int y_offset = base_y + iy;

      if(y_offset >= height || y_offset < 0)continue; // check if the offset index over y is valid (not larger than or equal to the size of the dimension) OR smaller than 0 (for fool proofing)

      for(int ix=0; ix<kernel_w; ix++){
        const int x_offset = base_x + ix;

        if(x_offset >= width || x_offset < 0)continue; // check if the offset index over x is valid (not larger than or equal to the size of the dimension) OR smaller than 0 (for fool proofing)

        const int offset = y_offset*width + x_offset;

        // Use this for verbose when debugging
        // printf("(ph: %d, pw: %d), base_y: %d, base_x: %d, iy: %d, ix: %d offset: %d \n", ph, pw, base_y, base_x, iy, ix, offset)

        mask_sum_max += exp(offset_bottom_input[offset]);

      }
    }
    // Overflow check
    mask_sum_max = clamp(mask_sum_max, lower, upper);


    for(int iy=0; iy<kernel_h; iy++){
      const int y_offset = base_y + iy; // offset adjustment (y-based)

      if(y_offset >= height || y_offset < 0)continue;

      for(int ix=0; ix<kernel_w; ix++){
        const int x_offset = base_x + ix; // offset adjustment (x-based)

        if(x_offset >= width || x_offset < 0)continue;
        const int offset = y_offset*width + x_offset; // x+y adjusted offset

        scalar_t mask_ = exp(offset_bottom_input[offset])/  mask_sum_max; // SoftMax

        output_data[index] += offset_bottom_input[offset] * mask_;
        output_data[index] = clamp(output_data[index], zero, upper);
      }
    }
  }
}


template <typename scalar_t>
__global__ void SoftPool3dForward(const int nthreads,
                                  const scalar_t *bottom_input, const int batches,
                                  const int channels, const int depth,
                                  const int height, const int width,
                                  const int kernel_d, const int kernel_h,
                                  const int kernel_w, const int stride_d,
                                  const int stride_h, const int stride_w,
                                  scalar_t *output_data){
    int pooled_depth = depth/stride_d;
    int pooled_height = height/stride_h;
    int pooled_width = width/stride_w;
    CUDA_1D_KERNEL_LOOP(index, nthreads) {
      int pw = index % pooled_width;
      int ph = (index / pooled_width) % pooled_height;
      int pd = (index / pooled_width / pooled_height) % pooled_depth;
      int c = (index / pooled_width / pooled_height / pooled_depth) % channels;
      int n = index / pooled_width / pooled_height / pooled_depth / channels;

      const int offset = (n * channels + c) * depth * height * width;
      const scalar_t *offset_bottom_input = bottom_input + offset;

      scalar_t mask_sum = 0.;
      output_data[index] = 0.;
      const scalar_t upper = n_limits<scalar_t>::max();
      const scalar_t lower = n_limits<scalar_t>::min();
      const scalar_t zero = 0.;

      for(int id=0; id<kernel_d; id++){
        const int d_offset = pd*stride_d + id - kernel_d/2;
        if(d_offset >= depth || d_offset < 0)continue;
        for(int iy=0; iy<kernel_h; iy++){
          const int y_offset = ph*stride_h + iy - kernel_h/2;
          if(y_offset >= height || y_offset < 0)continue;
          for(int ix=0; ix<kernel_w; ix++){
            const int x_offset = pw*stride_w + ix - kernel_w/2;
            if(x_offset >= width || x_offset < 0)continue;
            const int offset = d_offset*height + y_offset*width + x_offset;

            // (Over/Under)flow check (A.) 0 <= e^{inp[offset]} <= FLT_MAX
            scalar_t mask = exp(offset_bottom_input[offset]);
            mask = clamp(mask, zero, upper);
            mask_sum += mask;
          }
        }
      }
      // Overflow check (B.) FLT_MIN <= sum{e^{inp[offset]}} <= FLT_MAX
      mask_sum = clamp(mask_sum, lower, upper);

      for(int id=0; id<kernel_d; id++){
        const int d_offset = pd*stride_d + id - kernel_d/2;
        if(d_offset >= depth || d_offset < 0)continue;
        for(int iy=0; iy<kernel_h; iy++){
          const int y_offset = ph*stride_h + iy - kernel_h/2;
          if(y_offset >= height || y_offset < 0)continue;
          for(int ix=0; ix<kernel_w; ix++){
            const int x_offset = pw*stride_w + ix - kernel_w/2;
            if(x_offset >= width || x_offset < 0)continue;
            const int offset = d_offset*height + y_offset*width + x_offset;

            // (Over/Under)flow check (C.) 0 <= e^{inp[offset]} <= FLT_MAX
            scalar_t mask = exp(offset_bottom_input[offset]);
            mask = clamp(mask, zero, upper);

            // Underflow check (D.) 0 <= e^{inp[offset]}/sum{e^{inp[offset]}} <= 1
            mask /=  mask_sum;
            mask = clamp(mask, zero, upper);

            // Underflow check (E.) 0 <= (e^{inp[offset]}/sum{e^{inp[offset]}}) * inp[offset] <= FLT_MAX
            scalar_t weighted_inp = offset_bottom_input[offset] * mask;
            weighted_inp = clamp(weighted_inp, zero, upper);

            // Overflow check (F.) 0 <= sum[(e^{inp[offset]}/sum{e^{inp[offset]}}) * inp[offset]] <= FLT_MAX
            output_data[index] += weighted_inp;
            output_data[index] = clamp(output_data[index], zero, upper);
          }
        }
      }
    }
}


int SoftPool1dForwardLauncher(const at::Tensor input, const int batches,
                             const int channels, const int dim,
                             const int kernel_d, const int stride_d,
                             at::Tensor output){
    const int output_size = batches * dim/stride_d * channels;
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
        input.scalar_type(), "SoftPool1dLauncherForward", ([&] {
        const scalar_t *bottom_input = input.data_ptr<scalar_t>();
        scalar_t *output_data = output.data_ptr<scalar_t>();

        SoftPool1dForward<scalar_t>
        <<<GET_BLOCKS(output_size), THREADS_PER_BLOCK>>>(
          output_size, bottom_input,
          batches, channels,
          dim, kernel_d,
          stride_d, output_data);
        })
      );

    hipError_t err = hipGetLastError();
    if (hipSuccess != err) {
      fprintf(stderr, "cudaCheckError() failed : %s\n", hipGetErrorString(err));
      exit(-1);
    }
  return 1;
}

int SoftPool2dForwardLauncher(const at::Tensor input, const int batches,
                             const int channels, const int height,
                             const int width, const int kernel_h,
                             const int kernel_w, const int stride_h,
                             const int stride_w, at::Tensor output){
    const int output_size = batches * height/stride_h * width/stride_w * channels;
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
        input.scalar_type(), "SoftPool2dLauncherForward", ([&] {
        const scalar_t *bottom_input = input.data_ptr<scalar_t>();
        scalar_t *output_data = output.data_ptr<scalar_t>();

        SoftPool2dForward<scalar_t>
        <<<GET_BLOCKS(output_size), THREADS_PER_BLOCK>>>(
          output_size, bottom_input,
          batches, channels,
          height, width,
          kernel_h, kernel_w,
          stride_h, stride_w,
          output_data);
        })
      );

    hipError_t err = hipGetLastError();
    if (hipSuccess != err) {
      fprintf(stderr, "cudaCheckError() failed : %s\n", hipGetErrorString(err));
      exit(-1);
    }
  return 1;
}

int SoftPool3dForwardLauncher(const at::Tensor input, const int batches,
                             const int channels, const int depth,
                             const int height, const int width,
                             const int kernel_d, const int kernel_h,
                             const int kernel_w, const int stride_d,
                             const int stride_h, const int stride_w,
                            at::Tensor output){
    const int output_size = batches * depth/stride_d * height/stride_h * width/stride_w * channels;
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
        input.scalar_type(), "SoftPool3dLauncherForward", ([&] {
        const scalar_t *bottom_input = input.data_ptr<scalar_t>();
        scalar_t *output_data = output.data_ptr<scalar_t>();

        SoftPool3dForward<scalar_t>
        <<<GET_BLOCKS(output_size), THREADS_PER_BLOCK>>>(
          output_size, bottom_input,
          batches, channels,
          depth, height,
          width, kernel_d,
          kernel_h, kernel_w,
          stride_d, stride_h,
          stride_w, output_data);
        })
      );

    hipError_t err = hipGetLastError();
    if (hipSuccess != err) {
      fprintf(stderr, "cudaCheckError() failed : %s\n", hipGetErrorString(err));
      exit(-1);
    }
  return 1;
}


template <typename scalar_t>
__global__ void SoftPool1dBackward(const int nthreads,
                              const scalar_t *diff_output, const scalar_t *data_input,
                              const int batches, const int channels,
                              const int dim, const int kernel_d,
                              const int stride_d, scalar_t *diff_input){
    int pooled_dim = dim/stride_d;
    // Run in parallel for each cell within each kernel region
    CUDA_1D_KERNEL_LOOP(index, nthreads) {
      int pd = index % pooled_dim; // index of each kernel operation in relation to the position in the input
      int c = (index / pooled_dim) % channels;
      int n = index / pooled_dim / channels;

      const int offset0 = (n * channels + c) * dim; // initial offset
      const scalar_t *offset_data_input = data_input + offset0; // offset based on the input data

      const scalar_t diff_output_index = diff_output[index]; // offset based on the output gradients
      scalar_t *offset_diff_input = diff_input + offset0; // offset based on the input gradients

      const int base_d = pd*stride_d; // start cell index for each kernel

      // --- Initialisations happen here ----
      scalar_t mask_sum_max = 0.;
      const scalar_t upper = n_limits<scalar_t>::max();
      const scalar_t lower = n_limits<scalar_t>::min();

      // Iterate over inputs cells within each kernel region in the input
      for(int id=0; id<kernel_d; id++){
        const int d_offset = base_d + id;

        if(d_offset >= dim || d_offset < 0)continue; // check if the offset index is valid (not larger than or equal to the size of the dimension) OR smaller than 0 (for fool proofing)
        const int offset = d_offset;

        // Use this for verbose when debugging
        //printf("(pd: %d), base_d: %d, id: %d, d_offset: %d \n", pd, base_d, id, d_offset);

        mask_sum_max += exp(offset_data_input[offset]);

      }
      // Overflow check
      mask_sum_max = clamp(mask_sum_max, lower, upper);

      for(int id=0; id<kernel_d; id++){
        const int d_offset = base_d + id;

        if(d_offset >= dim || d_offset < 0)continue;
          const int offset = d_offset;

          scalar_t mask_ = exp(offset_data_input[offset])/mask_sum_max; // SoftMax

          scalar_t weighted_grad = diff_output_index * mask_; // use mask over the output gradients

          // Underflow check
          weighted_grad = clamp(weighted_grad, lower, upper);

          atomicAdd(offset_diff_input+offset, weighted_grad);
      }
    }
}

template <typename scalar_t>
__global__ void SoftPool2dBackward(const int nthreads,
                              const scalar_t *diff_output, const scalar_t *data_input,
                              const int batches, const int channels,
                              const int height, const int width,
                              const int kernel_h, const int kernel_w,
                              const int stride_h, const int stride_w,
                              scalar_t *diff_input){
    int pooled_height = height/stride_h;
    int pooled_width = width/stride_w;
    // Run in parallel for each cell within each kernel region
    CUDA_1D_KERNEL_LOOP(index, nthreads) {
      int pw = index % pooled_width; // index over width of each kernel operation in relation to the position in the input
      int ph = (index / pooled_width) % pooled_height; // index  over height of each kernel operation in relation to the position in the input
      int c = (index / pooled_width / pooled_height) % channels;
      int n = index / pooled_width / pooled_height / channels;

      const int offset0 = (n * channels + c) * height * width; // initial offset
      const scalar_t *offset_data_input = data_input + offset0; // offset based on the input data

      const scalar_t diff_output_index = diff_output[index]; // offset based on the output gradients
      scalar_t *offset_diff_input = diff_input + offset0; // offset based on the input gradients

      const int base_y = ph * stride_h; // start cell index over height/y for each kernel
      if (base_y > height - kernel_h)break; // limit height/y iterations for the index of the final kernel location in the input

      const int base_x = pw * stride_w; // start cell index over width/x for each kernel
      if (base_x > width - kernel_w)break; // limit width/x iterations for the index of the final kernel location in the input

      // --- Initialisations happen here ----
      scalar_t mask_sum_max = 0.;

      const scalar_t upper = n_limits<scalar_t>::max();
      const scalar_t lower = n_limits<scalar_t>::min();

      // Iterate over inputs cells within each kernel region in the input
      for(int iy=0; iy<kernel_h; iy++){
        const int y_offset = base_y + iy;

        if(y_offset >= height || y_offset < 0)continue; // check if the offset index over y is valid (not larger than or equal to the size of the dimension) OR smaller than 0 (for fool proofing)

        for(int ix=0; ix<kernel_w; ix++){
          const int x_offset = base_x + ix;

          if(x_offset >= width || x_offset < 0)continue; // check if the offset index over x is valid (not larger than or equal to the size of the dimension) OR smaller than 0 (for fool proofing)

          const int offset = y_offset*width + x_offset;

          // Use this for verbose when debugging
          // printf("(ph: %d, pw: %d), base_y: %d, base_x: %d, iy: %d, ix: %d offset: %d \n", ph, pw, base_y, base_x, iy, ix, offset)

          mask_sum_max += exp(offset_data_input[offset]);

        }
      }
      // Overflow check
      mask_sum_max = clamp(mask_sum_max, lower, upper);

      for(int iy=0; iy<kernel_h; iy++){
        const int y_offset = base_y + iy; // offset adjustment (y-based)

        if(y_offset >= height || y_offset < 0)continue;
        for(int ix=0; ix<kernel_w; ix++){
          const int x_offset = base_x + ix;

          if(x_offset >= width || x_offset < 0)continue;
            const int offset = y_offset*width + x_offset; // offset adjustment (x-based)

            scalar_t mask_ = exp(offset_data_input[offset])/mask_sum_max; // SoftMax (sum)

            scalar_t weighted_grad = diff_output_index * mask_; // use mask over the output gradients

            // Underflow check
            weighted_grad = clamp(weighted_grad, lower, upper);

            atomicAdd(offset_diff_input+offset, weighted_grad);
        }
      }
    }
}

template <typename scalar_t>
__global__ void SoftPool3dBackward(const int nthreads,
                              const scalar_t *diff_output, const scalar_t *data_input,
                              const int batches, const int channels,
                              const int depth, const int height,
                              const int width, const int kernel_d,
                              const int kernel_h, const int kernel_w ,
                              const int stride_d, const int stride_h,
                              const int stride_w, scalar_t *diff_input){
    int pooled_depth = depth/stride_d;
    int pooled_height = width/stride_h;
    int pooled_width = width/stride_w;
    CUDA_1D_KERNEL_LOOP(index, nthreads) {
      int pw = index % pooled_width; // index over width of each kernel operation in relation to the position in the input
      int ph = (index / pooled_width) % pooled_height; // index over height of each kernel operation in relation to the position in the input
      int pd = (index / pooled_width / pooled_height) % pooled_depth; // index over depth of each kernel operation in relation to the position in the input
      int c = (index / pooled_width / pooled_height / pooled_depth) % channels;
      int n = index / pooled_width / pooled_height / pooled_depth / channels;

      const int offset0 = (n * channels + c) * depth * height * width; // initial offset
      const scalar_t *offset_data_input = data_input + offset0; // offset based on the input data

      const scalar_t diff_output_index = diff_output[index]; // offset based on the output gradients
      scalar_t *offset_diff_input = diff_input + offset0; // offset based on the input gradients

      const int base_d = pd*stride_d; // start cell index over depth/d for each kernel
      if (base_d > depth - kernel_d)break; // limit depth/d iterations for the index of the final kernel location in the input

      const int base_y = ph*stride_h; // start cell index over height/y for each kernel
      if (base_y > height - kernel_h)break; // limit height/y iterations for the index of the final kernel location in the input

      const int base_x = pw*stride_w; // start cell index over width/x for each kernel
      if (base_x > width - kernel_w)break; // limit width/x iterations for the index of the final kernel location in the input

      // --- Initialisations happen here ----
      scalar_t mask_sum_max = 0.;

      const scalar_t upper = n_limits<scalar_t>::max();
      const scalar_t lower = n_limits<scalar_t>::min();

      // Iterate over inputs cells within each kernel region in the input
      for(int id=0; id<kernel_d; id++){
        const int d_offset = base_d + id;

        if(d_offset >= depth || d_offset < 0)continue; // check if the offset index over d is valid (not larger than or equal to the size of the dimension) OR smaller than 0 (for fool proofing)

        for(int iy=0; iy<kernel_h; iy++){
          const int y_offset = base_y + iy;

          if(y_offset >= height || y_offset < 0)continue; // check if the offset index over y is valid (not larger than or equal to the size of the dimension) OR smaller than 0 (for fool proofing)

          for(int ix=0; ix<kernel_w; ix++){
            const int x_offset = base_x + ix;

            if(x_offset >= width || x_offset < 0)continue; // check if the offset index over x is valid (not larger than or equal to the size of the dimension) OR smaller than 0 (for fool proofing)

            const int offset = d_offset*height + y_offset*width + x_offset;

            // Use this for verbose when debugging
            // printf("(pd: %d, ph: %d, pw: %d), base_d: %d, base_y: %d, base_x: %d, id: %d, iy: %d, ix: %d, offset: %d \n", pd, ph, pw, base_d, base_y, base_x, id, iy, ix, offset);

            mask_sum_max += exp(offset_data_input[offset]);

          }
        }
      }
      // Overflow check
      mask_sum_max = clamp(mask_sum_max, lower, upper);

      for(int id=0; id<kernel_d; id++){
        const int d_offset = base_d + id; // offset adjustment (d-based)

        if(d_offset >= depth || d_offset < 0)continue;
        for(int iy=0; iy<kernel_h; iy++){
          const int y_offset = base_y + iy; // offset adjustment (y-based)

          if(y_offset >= height || y_offset < 0)continue;
          for(int ix=0; ix<kernel_w; ix++){
            const int x_offset = base_x + ix; // offset adjustment (x-based)

            if(x_offset >= width || x_offset < 0)continue;
              const int offset = d_offset*height + y_offset*width + x_offset;

              scalar_t mask_ = exp(offset_data_input[offset])/mask_sum_max; // SoftMax

              scalar_t weighted_grad = diff_output_index * mask_; // use mask over the output gradients

              // Underflow check
              weighted_grad = clamp(weighted_grad, lower, upper);

              atomicAdd(offset_diff_input+offset, weighted_grad);
          }
        }
      }
    }
}

int SoftPool1dBackwardLauncher(const at::Tensor output_grad, const at::Tensor input,
                               const int batches, const int channels,
                               const int dim, const int kernel_d,
                               const int stride_d, at::Tensor input_grad){

    const int output_size = batches * dim/stride_d * channels;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
        input.scalar_type(), "SoftPool1dLauncherBackward", ([&] {
        scalar_t *diff_input = input_grad.data_ptr<scalar_t>();
        const scalar_t *diff_output = output_grad.data_ptr<scalar_t>();
        const scalar_t *data_input = input.data_ptr<scalar_t>();

        SoftPool1dBackward<scalar_t>
        <<<GET_BLOCKS(output_size), THREADS_PER_BLOCK>>>(
          output_size, diff_output,
          data_input, batches,
          channels, dim,
          kernel_d, stride_d,
          diff_input);
        }
        )
        );

    hipError_t err = hipGetLastError();
    if (hipSuccess != err) {
      fprintf(stderr, "cudaCheckError() failed : %s\n", hipGetErrorString(err));
      exit(-1);
    }
  return 1;
}

int SoftPool2dBackwardLauncher(const at::Tensor output_grad, const at::Tensor input,
                               const int batches, const int channels,
                               const int height, const int width,
                               const int kernel_h, const int kernel_w,
                               const int stride_h, const int stride_w,
                               at::Tensor input_grad){

    const int output_size = batches * height/stride_h * width/stride_w * channels;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
        input.scalar_type(), "SoftPool2dLauncherBackward", ([&] {
        scalar_t *diff_input = input_grad.data_ptr<scalar_t>();
        const scalar_t *diff_output = output_grad.data_ptr<scalar_t>();
        const scalar_t *data_input = input.data_ptr<scalar_t>();

        SoftPool2dBackward<scalar_t>
        <<<GET_BLOCKS(output_size), THREADS_PER_BLOCK>>>(
          output_size, diff_output,
          data_input, batches,
          channels, height,
          width, kernel_h,
          kernel_w, stride_h,
          stride_w, diff_input);
        }
        )
        );

    hipError_t err = hipGetLastError();
    if (hipSuccess != err) {
      fprintf(stderr, "cudaCheckError() failed : %s\n", hipGetErrorString(err));
      exit(-1);
    }
  return 1;
}

int SoftPool3dBackwardLauncher(const at::Tensor output_grad, const at::Tensor input,
                               const int batches, const int channels,
                               const int depth, const int height,
                               const int width, const int kernel_d,
                               const int kernel_h, const int kernel_w,
                               const int stride_d, const int stride_h,
                               const int stride_w, at::Tensor input_grad){

    const int output_size = batches * depth/stride_d * height/stride_h * width/stride_w * channels;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
        input.scalar_type(), "SoftPool3dLauncherBackward", ([&] {
        scalar_t *diff_input = input_grad.data_ptr<scalar_t>();
        const scalar_t *diff_output = output_grad.data_ptr<scalar_t>();
        const scalar_t *data_input = input.data_ptr<scalar_t>();

        SoftPool3dBackward<scalar_t>
        <<<GET_BLOCKS(output_size), THREADS_PER_BLOCK>>>(
          output_size, diff_output,
          data_input, batches,
          channels, depth, height,
          width, kernel_d,
          kernel_h, kernel_w,
          stride_d, stride_h,
          stride_w, diff_input);
        }
        )
        );

    hipError_t err = hipGetLastError();
    if (hipSuccess != err) {
      fprintf(stderr, "cudaCheckError() failed : %s\n", hipGetErrorString(err));
      exit(-1);
    }
  return 1;
}
